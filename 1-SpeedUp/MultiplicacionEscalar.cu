#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

/*
 * Implementación de un programa en CUDA que realiza la multiplicación de un vector por un escalar
 * de forma paralela, comparando los resultados obtenidos en la CPU y la GPU, y obteniendo el SpeedUp.
 *
 * @author: Ayala Morales Mauricio
 */

#define N 9000000

/*
 * Función que inicializa un arreglo con números aleatorios entre 1 y 100.
 */
void initialize(float *A, int size)
{
    srand(0);

    for (int i = 0; i < size; i++)
    {
        A[i] = rand() % 100 + 1;
    }
}

/*
 * Función que realiza la multiplicación de un vector (arreglo) de números aleatorios
 * por un escalar, utilizando la CPU.
 */
void multOnCPU(float *A, float c, float *C, int size)
{
    for (int i = 0; i < size; i++)
    {
        C[i] = A[i] * c;
    }
}

/*
 * 4. Kernel en CUDA que realiza la multiplicación de un vector (arreglo) de números aleatorios
 * por un escalar, utilizando la GPU.
 */
__global__ void multOnGPU(float *A, float c, float *C, int size)
{
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx < size)
    {
        C[idx] = A[idx] * c;
    }
}

/*
 * Función que obtiene la hora en segundos.
 */
double cpuTime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

/*
 * Función que valida los resultados de la multiplicación, obtenidos en la GPU y CPU.
 */
void validate(float *res_GPU, float *res_CPU, int size)
{
    double epsilon = 1.0E-8;
    for (int i = 0; i < size; i++)
    {
        if (abs(res_GPU[i] - res_CPU[i]) > epsilon)
        {
            printf("ERROR: producto distinto\n");
            return;
        }
    }

    printf("PRODUCTO CORRECTO\n");
}

/*
 * Función que calcula el SpeedUp.
 */
double speedUp(double h_time, double d_time)
{
    return h_time / d_time;
}

int main()
{

    // 2. Asignación de memoria dinámica
    // -------------- CPU --------------

    size_t nBytes = N * sizeof(float);
    float *h_A, *h_C, *h_res;

    h_A = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);
    h_res = (float *)malloc(nBytes);

    float *d_A, *d_C;

    //--------------- GPU --------------
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_C, nBytes);
    //----------------------------------

    // 1. Inicialización de datos
    initialize(h_A, N);
    float c = rand() % 100 + 1;

    memset(h_C, 0, nBytes);
    memset(h_res, 0, nBytes);
    // --------------------------

    // 3. Transferencia de datos, del Host al Device (CPU a GPU)
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, nBytes, hipMemcpyHostToDevice);
    //----------------------------------------------------------

    // 5. Configuración de la ejecución en la GPU
    int b = (N / 1024) + 1;
    dim3 block(b);
    dim3 thread(1024);
    double tic = cpuTime();
    multOnGPU<<<block, thread>>>(d_A, c, d_C, N);
    hipDeviceSynchronize();
    double toc = cpuTime();
    double tictocGPU = toc - tic;
    printf("GPU time: %lf segs.\n", tictocGPU);
    //-------------------------------------------

    // 3. Transferencia de datos, del Device al Host (GPU a CPU)
    hipMemcpy(h_res, d_C, nBytes, hipMemcpyDeviceToHost);
    //----------------------------------------------------------

    tic = cpuTime();
    multOnCPU(h_A, c, h_C, N);
    toc = cpuTime();
    double tictocCPU = toc - tic;
    printf("CPU time: %lf segs.\n", tictocCPU);

    // 7. Validación de los resultados
    validate(h_res, h_C, N);
    //-------------------------------

    // 8. Liberación de memoria
    // --------- CPU ----------
    free(h_A);
    free(h_C);

    // --------- GPU ----------
    hipFree(d_A);
    hipFree(d_C);
    //-------------------------

    // 9. SpeedUp
    printf("SpeedUp: %lf\n", speedUp(tictocCPU, tictocGPU));

    return 0;
}