#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define Nx 5
#define Ny 3
#define Mx 3
#define My 7

/**
 * Función que inicializa una matriz con números aleatorios entre 1 y 100.
 *
 * @param x: Cantidad de filas de la matriz.
 * @param y: Cantidad de columnas de la matriz.
 * @param c: Si c == 0, se le asigna 0 a cada elemento de la matriz, si c != 0 se asigna un número
 * aleatorio.
 */
float **initialize(int x, int y, short c)
{
    float **A = (float **)malloc(x * sizeof(float *));

    for (int i = 0; i < x; i++)
    {
        A[i] = (float *)malloc(y * sizeof(float));
        for (int j = 0; j < y; j++)
        {
            if (c == 0)
            {
                A[i][j] = 0;
                break;
            }

            A[i][j] = rand() % 100 + 1;
        }
    }
    return A;
}

/**
 * Función que realiza la conversión de una matriz a un arreglo, es decir, de un arreglo de 2
 * dimensiones a un arreglo de 1 dimensión.
 *
 * @param A: Matriz a convertir.
 * @param V: Vector donde se almacenará la matriz.
 * @param x: Cantidad de filas de la matriz.
 * @param y: Cantidad de columnas de la matriz.
 */
void matrixToVector(float **A, float *V, int x, int y)
{
    for (int i = 0; i < x; i++)
    {
        for (int j = 0; j < y; j++)
        {
            V[(y * i) + j] = A[i][j];
        }
    }
}

/**
 * 4. Kernel en CUDA que realiza el producto de dos matrices en su representación vectorial, de
 * números aleatorios utilizando la GPU.  
 *
 * @param A: Primer matriz a multiplicar, en forma de arreglo.
 * @param B: Segunda matriz a multiplicar, en forma de arreglo.
 * @param C: Arreglo donde se almacenará el resultado del pructo.
 * @param ax: Cantidad de filas de la matriz A.
 * @param ay: Cantidad de columnas de la matriz A.
 * @param bx: Cantidad de filas de la matriz B.
 * @param by: Cantidad de columnas de la matriz B.
 */
__global__ void prodOnGPU(float *A, float *B, float *C, int ax, int ay, int bx, int by)
{
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    int idy = threadIdx.y + (blockIdx.y * blockDim.y);
    int index = idy + (idx * ax);

    if (idx < ax && idy < by)
    {
        for (int i = 0; i < ay; i++)
        {
            C[index] += A[(idx * ay) + i] * B[idy + (by * i)];
        }
    }
}

/**
 * Función que calcula el producto de dos matrices, en CPU.
 *
 * @param A: Matriz A.
 * @param B: Matriz B.
 * @param C: Matriz donde se almacenará el resultado de A x B.
 */
void prodOnCPU(float **A, float **B, float **C, int ax, int ay, int bx, int by)
{
    if (ay != bx)
    {
        printf("ERROR: La cantidad de columnas de A debe ser igual a la cantidad de filas de B\n");
        return;
    }
    for (int i = 0; i < ax; i++)
    {
        for (int j = 0; j < by; j++)
        {
            for (int k = 0; k < ay; k++)
            {
                C[i][j] += A[i][k] * B[k][j];
            }
        }
    }
}

/**
 * Función que obtiene la hora en segundos.
 *
 * @returns El tiempo actual del cpu en segundos.
 */
double cpuTime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-06);
}

/**
 * Función que valida los resultados del producto, obtenidos en la GPU y CPU.
 *
 * @param res_GPU: Resultado del producto obtenido por la GPU, en forma de vector.
 * @param res_CPU: Resultado del producto obtenido por la CPU, en forma matricial.
 * @param x: Cantidad de filas de la matriz.
 * @param y: Cantidad de columnas de la matriz.
 */
void validate(float *res_GPU, float **res_CPU, int x, int y)
{
    double epsilon = 1.0E-8;
    for (int i = 0; i < x; i++)
    {
        for (int j = 0; j < y; j++)
        {
            if (abs(res_GPU[(x * i) + j] - res_CPU[i][j]) > epsilon)
            {
                printf("ERROR: producto distinto\n");
                return;
            }
        }
    }

    printf("PRODUCTO CORRECTO\n");
}

void validateMatrices(int ax, int ay, int bx, int by){
    if(ay != bx){
        printf("ERROR: La cantidad de columnas de A debe ser igual a la cantidad de filas de B\n");
        exit(1);
    }
}

/**
 * Función que calcula el SpeedUp.
 *
 * @param h_time: Tiempo utilizado por el Host (CPU) para realizar la operación.
 * @param d_time: Tiempo utilizado por el Host (GPU) para realizar la operación.
 *
 * @returns El SpeedUp obtenido del algoritmo ejecutado en paralelo en comparación con su ejecución
 * en serie en una sola unidad de procesamiento.
 */
double speedUp(double h_time, double d_time)
{
    return h_time / d_time;
}

void printMatrix(float **A, int x, int y){
    for(int i=0; i<x; i++){
        printf("[");
        for(int j=0; j<y; j++){
            printf("%.2f ", A[i][j]);
        }
        printf("]\n");
    }
}

void printVector(float *A, int size){
    printf("[");
    for(int i=0; i < size; i++){
        printf("%.2f ", A[i]);
    }
    printf("]\n");
}

int main()
{

    // 1. Inicialización de datos
    // -------------- CPU --------------
    float **h_A, **h_B, **h_C, *h_VA, *h_VB, *h_res;

    float *d_A, *d_B, *d_C;
    // ---------------------------------

    validateMatrices(Nx, Ny, Mx, My);

    /// 2. Asignación de memoria dinámica
    // ----------------------------------
    h_A = initialize(Nx, Ny, 1);
    h_B = initialize(Mx, My, 1);
    h_C = initialize(Nx, My, 0);
    h_VA = new float[Nx * Ny];
    h_VB = new float[Mx * My];
    h_res = new float[Nx * My];

    matrixToVector(h_A, h_VA, Nx, Ny);
    matrixToVector(h_B, h_VB, Mx, My);
    matrixToVector(h_C, h_res, Nx, My);
    //--------------- GPU ---------------
    hipMalloc((float **)&d_A, Nx * Ny * sizeof(float));
    hipMalloc((float **)&d_B, Mx * My * sizeof(float));
    hipMalloc((float **)&d_C, Nx * My * sizeof(float));
    // ----------------------------------

    // 3. Transferencia de datos, del Host al Device (CPU a GPU)
    hipMemcpy(d_A, h_VA, Nx * Ny * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_VB, Mx * My * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_res, Nx * My * sizeof(float), hipMemcpyHostToDevice);
    //----------------------------------------------------------

    double tic, toc, timeCPU, timeGPU;

    // 5. Configuración de la ejecución en la GPU.
    // -------------------------------------------

    int bx = (int)(Nx / 32) + 1;
    int by = (int)(My / 32) + 1;
    dim3 grid(bx, by);

    int tx = (Nx <= 32)?  Nx : 32;
    int ty = (My <= 32)? My : 32;
    printf("Block: %d x %d\n", bx, by);
    printf("Thread: %d x %d\n", tx, ty);
    dim3 block(tx, ty);

    tic = cpuTime();
    prodOnGPU<<<grid, block>>>(d_A, d_B, d_C, Nx, Ny, Mx, My);
    hipDeviceSynchronize();
    toc = cpuTime();
    timeGPU = toc - tic;
    printf("GPU time: %lf segs.\n", timeGPU);
    // -------------------------------------------

    // 6. Transferencia de datos, del Device al Host (GPU a CPU)
    hipMemcpy(h_res, d_C, Nx * My * sizeof(float), hipMemcpyDeviceToHost);
    //----------------------------------------------------------

    // Producto de matrices en CPU.
    // -------------------------------
    tic = cpuTime();
    prodOnCPU(h_A, h_B, h_C, Nx, Ny, Mx, My);
    toc = cpuTime();
    timeCPU = toc - tic;
    printf("CPU time: %lf segs.\n", timeCPU);
    // -------------------------------

    printVector(h_res, Nx * My);
    printMatrix(h_C, Nx, My);

    // 7. Validación de resultados.
    // ------------------------------
    validate(h_res, h_C, Nx, Ny);
    // ------------------------------

    // 8. Liberación de memoria.
    // ----------------------
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_VA);
    free(h_VB);
    free(h_res);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // ----------------------

    // 9. SpeedUp.
    // -----------
    printf("SpeedUp : %lf\n", speedUp(timeCPU, timeGPU));
    // -----------

    return 0;
}